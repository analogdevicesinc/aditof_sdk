#include "hip/hip_runtime.h"
/*
 * BSD 3-Clause License
 *
 * Copyright (c) 2019, Analog Devices, Inc.
 * All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions are met:
 *
 * 1. Redistributions of source code must retain the above copyright notice, this
 *    list of conditions and the following disclaimer.
 *
 * 2. Redistributions in binary form must reproduce the above copyright notice,
 *    this list of conditions and the following disclaimer in the documentation
 *    and/or other materials provided with the distribution.
 *
 * 3. Neither the name of the copyright holder nor the names of its
 *    contributors may be used to endorse or promote products derived from
 *    this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS"
 * AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE
 * DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE LIABLE
 * FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL
 * DAMAGES (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR
 * SERVICES; LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER
 * CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY,
 * OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */
#include "../include/cuda_utils.h"
#include <assert.h>
#include <iostream>
#include <stdio.h>
namespace aditof {};

// -------------------      CUDA        -----------------------------------------------

// Convenience function for checking CUDA runtime API results
// can be wrapped around any runtime API call. No-op in release builds.
inline hipError_t checkCuda(hipError_t result) {
#if defined(DEBUG) || defined(_DEBUG)
    if (result != hipSuccess) {
        fprintf(stderr, "CUDA_CXX: DA Runtime Error: %s\n",
                hipGetErrorString(result));
        assert(result == hipSuccess);
    }
#endif
    return result;
}

__global__ void buildDistortionCorrectionCacheCuda(double *m_distortion_cache_d,
                                                   double *m_parameters_d) {

    int threadPosition = blockIdx.x * THREAD_PER_BLOCK + threadIdx.x;

    int i = threadPosition % (int)m_parameters_d[0];
    int j = threadPosition / (int)m_parameters_d[0];

    if (i >= 0 && i < m_parameters_d[0]) {
        if (j >= 0 && j < m_parameters_d[1]) {
            double x = (i - m_parameters_d[4]) / m_parameters_d[2];
            double y = (j - m_parameters_d[5]) / m_parameters_d[3];

            double r2 = x * x + y * y;
            double k_calc = double(1 + m_parameters_d[6] * r2 +
                                   m_parameters_d[7] * r2 * r2 +
                                   m_parameters_d[8] * r2 * r2 * r2);
            m_distortion_cache_d[j * (int)m_parameters_d[0] + i] = k_calc;
        }
    }
}

__global__ void
applyDistortionCorrectionCacheCuda(uint16_t *m_frame_d, uint16_t *tmp_frame,
                                   double *m_parameters_d,
                                   double *m_distortion_cache_d) {

    int threadPosition = blockIdx.x * THREAD_PER_BLOCK + threadIdx.x;

    int i = threadPosition % (int)m_parameters_d[0];
    int j = threadPosition / (int)m_parameters_d[0];

    if (i >= 0 && i < m_parameters_d[0]) {
        if (j >= 0 && j < m_parameters_d[1]) {

            double x = (double(i) - m_parameters_d[4]) / m_parameters_d[2];
            double y = (double(j) - m_parameters_d[5]) / m_parameters_d[3];

            //apply correction
            double x_dist_adim =
                x * m_distortion_cache_d[j * (int)m_parameters_d[0] + i];
            double y_dist_adim =
                y * m_distortion_cache_d[j * (int)m_parameters_d[0] + i];

            //back to original space
            int x_dist =
                (int)(x_dist_adim * m_parameters_d[2] + m_parameters_d[4]);
            int y_dist =
                (int)(y_dist_adim * m_parameters_d[3] + m_parameters_d[5]);

            if (x_dist >= 0 && x_dist < (int)m_parameters_d[0] && y_dist >= 0 &&
                y_dist < (int)m_parameters_d[1]) {
                m_frame_d[j * (int)m_parameters_d[0] + i] =
                    tmp_frame[y_dist * (int)m_parameters_d[0] + x_dist];
            } else {
                m_frame_d[j * (int)m_parameters_d[0] + i] =
                    tmp_frame[j * (int)m_parameters_d[0] + i];
            }
        }
    }
}

__global__ void buildGeometryCorrectionCacheCuda(double *m_geometry_cache_d,
                                                 double *m_parameters_d) {

    int threadPosition = blockIdx.x * THREAD_PER_BLOCK + threadIdx.x;

    int i = threadPosition / (int)m_parameters_d[0];
    int j = threadPosition % (int)m_parameters_d[0];

    if (i >= 0 && i < m_parameters_d[1]) {
        if (j >= 0 && j < m_parameters_d[0]) {
            double tanXAngle = (m_parameters_d[9] - j) / m_parameters_d[2];
            double tanYAngle = (m_parameters_d[10] - i) / m_parameters_d[3];

            m_geometry_cache_d[i * (int)m_parameters_d[0] + j] =
                1.0 / sqrt(1 + tanXAngle * tanXAngle + tanYAngle * tanYAngle);
        }
    }
}

__global__ void applyGeometryCorrectionCacheCuda(uint16_t *m_frame_d,
                                                 double *m_parameters_d,
                                                 double *m_geometry_cache_d

) {
    int threadPosition = blockIdx.x * THREAD_PER_BLOCK + threadIdx.x;

    if (threadPosition >= 0 &&
        threadPosition < m_parameters_d[0] * m_parameters_d[1]) {
        if (m_frame_d[threadPosition] > m_parameters_d[14])
            m_frame_d[threadPosition] = m_parameters_d[14];
        else
            m_frame_d[threadPosition] =
                m_frame_d[threadPosition] * m_geometry_cache_d[threadPosition];
    }
}

__global__ void buildDepthCorrectionCacheCuda(uint16_t *m_depth_cache_d,
                                              double *m_parameters_d) {

    int threadPosition = blockIdx.x * THREAD_PER_BLOCK + threadIdx.x;
    if (threadPosition < m_parameters_d[13]) {
        int16_t currentValue = static_cast<int16_t>(
            static_cast<float>(threadPosition) * m_parameters_d[11] +
            m_parameters_d[12]);
        m_depth_cache_d[threadPosition] = currentValue <= m_parameters_d[14]
                                              ? currentValue
                                              : m_parameters_d[14];
    }
}

__global__ void applyDepthCorrectionCacheCuda(uint16_t *m_frame_d,
                                              double *m_parameters_d,
                                              uint16_t *m_depth_cache_d) {

    int threadPosition = blockIdx.x * THREAD_PER_BLOCK + threadIdx.x;

    if (threadPosition >= 0 &&
        threadPosition < m_parameters_d[0] * m_parameters_d[1]) {
        *(m_frame_d + threadPosition) =
            *(m_depth_cache_d + *(m_frame_d + threadPosition));
    }
}

//--------------------------    CLASS   -------------------------------------------------------------

void cudaOnTarget::buildDistortionCorrectionCache() {

    std::cout << "CUDA_CXX: Building Distortion correction\n";

    checkCuda(hipMalloc((void **)&m_distortion_cache_d,
                         sizeof(double) * m_parameters[0] * m_parameters[1]));

    buildDistortionCorrectionCacheCuda<<<m_parameters[0] * m_parameters[1] /
                                             THREAD_PER_BLOCK,
                                         THREAD_PER_BLOCK>>>(
        m_distortion_cache_d, m_parameters_d);
}

void cudaOnTarget::buildGeometryCorrectionCache() {

    std::cout << "CUDA_CXX: Building Geometry correction\n";

    checkCuda(hipMalloc((void **)&m_geometry_cache_d,
                         sizeof(double) * m_parameters[0] * m_parameters[1]));

    //Check if more blocks nedded than resulted from division
    int nrOfBlocks =
        ((m_parameters[0] * m_parameters[1] / THREAD_PER_BLOCK) *
             THREAD_PER_BLOCK <
         m_parameters[0] * m_parameters[1])
            ? m_parameters[0] * m_parameters[1] / THREAD_PER_BLOCK + 1
            : m_parameters[0] * m_parameters[1] / THREAD_PER_BLOCK;
    buildGeometryCorrectionCacheCuda<<<nrOfBlocks, THREAD_PER_BLOCK>>>(
        m_geometry_cache_d, m_parameters_d);
}

void cudaOnTarget::buildDepthCorrectionCache() {

    std::cout << "CUDA_CXX: Building Depth correction\n";

    checkCuda(hipMalloc((void **)&m_depth_cache_d,
                         sizeof(uint16_t) * m_parameters[13]));

    //Check if more blocks nedded than resulted from division
    int nrOfBlocks = ((m_parameters[13] / THREAD_PER_BLOCK) * THREAD_PER_BLOCK <
                      m_parameters[13])
                         ? m_parameters[13] / THREAD_PER_BLOCK + 1
                         : m_parameters[13] / THREAD_PER_BLOCK;
    buildDepthCorrectionCacheCuda<<<nrOfBlocks, THREAD_PER_BLOCK>>>(
        m_depth_cache_d, m_parameters_d);
}

void cudaOnTarget::applyDistortionCorrection() {

    //create temporary frame buffer
    uint16_t *tmp_frame;
    checkCuda(hipMalloc((void **)&tmp_frame,
                         sizeof(uint16_t) * m_parameters[0] * m_parameters[1]));
    checkCuda(hipMemcpy(tmp_frame, m_frame_d,
                         sizeof(uint16_t) * m_parameters[0] * m_parameters[1],
                         hipMemcpyDeviceToDevice));

    //Check if more blocks nedded than resulted from division
    int nrOfBlocks =
        ((m_parameters[0] * m_parameters[1] / THREAD_PER_BLOCK) *
             THREAD_PER_BLOCK <
         m_parameters[0] * m_parameters[1])
            ? m_parameters[0] * m_parameters[1] / THREAD_PER_BLOCK + 1
            : m_parameters[0] * m_parameters[1] / THREAD_PER_BLOCK;
    applyDistortionCorrectionCacheCuda<<<nrOfBlocks, THREAD_PER_BLOCK>>>(
        m_frame_d, tmp_frame, m_parameters_d, m_distortion_cache_d);
    checkCuda(hipFree(tmp_frame));
}
void cudaOnTarget::applyDepthCorrection() {

    //Check if more blocks nedded than resulted from division
    int nrOfBlocks =
        ((m_parameters[0] * m_parameters[1] / THREAD_PER_BLOCK) *
             THREAD_PER_BLOCK <
         m_parameters[0] * m_parameters[1])
            ? m_parameters[0] * m_parameters[1] / THREAD_PER_BLOCK + 1
            : m_parameters[0] * m_parameters[1] / THREAD_PER_BLOCK;
    applyDepthCorrectionCacheCuda<<<nrOfBlocks, THREAD_PER_BLOCK>>>(
        m_frame_d, m_parameters_d, m_depth_cache_d);
}

void cudaOnTarget::applyGeometryCorrection() {
    //Check if more blocks nedded than resulted from division
    int nrOfBlocks =
        ((m_parameters[0] * m_parameters[1] / THREAD_PER_BLOCK) *
             THREAD_PER_BLOCK <
         m_parameters[0] * m_parameters[1])
            ? m_parameters[0] * m_parameters[1] / THREAD_PER_BLOCK + 1
            : m_parameters[0] * m_parameters[1] / THREAD_PER_BLOCK;
    applyGeometryCorrectionCacheCuda<<<nrOfBlocks, THREAD_PER_BLOCK>>>(
        m_frame_d, m_parameters_d, m_geometry_cache_d);
}

void cudaOnTarget::cpyFrameToGPU(uint16_t *frame) {
    checkCuda(hipMemcpy(m_frame_d, frame,
                         sizeof(uint16_t) * m_parameters[0] * m_parameters[1],
                         hipMemcpyHostToDevice));
    memcpy(m_frame, frame, 640 * 480 * sizeof(uint16_t));
}
void cudaOnTarget::cpyFrameFromGPU(uint16_t *frame) {
    checkCuda(hipMemcpy(frame, m_frame_d,
                         sizeof(uint16_t) * m_parameters[0] * m_parameters[1],
                         hipMemcpyDeviceToHost));
}

void cudaOnTarget::printFrameFromGPU() {
    cpyFrameFromGPU(m_frame);
    for (int i = 0; i < 10; i++) {
        std::cout << m_frame[i] << ", ";
    }
}

void cudaOnTarget::setParameters(double *parameters) {

    //Parameter order:
    // width_tmp, height_tmp, fx_tmp, fy_tmp, cx_tmp, cy_tmp,
    //                           k1_tmp, k2_tmp, 0, x0_tmp, y0_tmp, gain_tmp,
    //                           offset_tmp, pixelMaxValue_tmp, range_tmp

    //Moving parameters on GPU memory
    m_parameters = (double *)malloc(15 * sizeof(double));
    memcpy(m_parameters, parameters, 15 * sizeof(double));

    checkCuda(hipMalloc((void **)&m_parameters_d, sizeof(double) * 15));
    checkCuda(hipMemcpy(m_parameters_d, parameters, sizeof(double) * 15,
                         hipMemcpyHostToDevice));

    //allocating memory for frame
    checkCuda(hipMalloc((void **)&m_frame_d,
                         sizeof(uint16_t) * m_parameters[0] * m_parameters[1]));
    m_frame = (uint16_t *)malloc(sizeof(uint16_t) * m_parameters[0] *
                                 m_parameters[1]);
}

void cudaOnTarget::freeAll() {
    checkCuda(hipFree(m_geometry_cache_d));
    checkCuda(hipFree(m_distortion_cache_d));
    checkCuda(hipFree(m_depth_cache_d));
    checkCuda(hipFree(m_frame_d));
    checkCuda(hipFree(m_parameters_d));
}
